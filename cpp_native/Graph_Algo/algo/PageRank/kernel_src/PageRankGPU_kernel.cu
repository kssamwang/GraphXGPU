#include "hip/hip_runtime.h"
#include "PageRankGPU_kernel.h"

__global__ void MSGApply_kernel(Vertex *vSet, double *vValues, int numOfMsg, int *mDstSet, PRA_MSG *mValueSet, double resetProb)
{
	int tid = threadIdx.x;

	if(tid < numOfMsg)
	{
        int vID = mDstSet[tid];

		//test
//		printf("vId : %d value : %f\n", vID, mValueSet[tid].rank);

		vSet[vID].isActive = true;
		vValues[(vID << 1) + 1] = (1.0 - resetProb) * mValueSet[tid].rank;
	}
}

hipError_t MSGApply_kernel_exec(Vertex *vSet, double *vValues, int numOfMsg, int *mDstSet, PRA_MSG *mValueSet, double resetProb)
{
	hipError_t err = hipSuccess;
	
	MSGApply_kernel<<<1, NUMOFGPUCORE>>>(vSet, vValues, numOfMsg, mDstSet, mValueSet, resetProb);
    err = hipGetLastError();

	hipDeviceSynchronize();
	
	return err;
}

__global__ void MSGGenMerge_kernel(PRA_MSG *mTransformdMergedMSGValueSet,
	Vertex *vSet, double *vValues, int numOfEdge, Edge *eSet)
{
	int tid = threadIdx.x;

	if(tid < numOfEdge)
	{
		int srcVid = eSet[tid].src;
		int mValueIndex = eSet[tid].dst;

        //test
//         printf("msg - srcVid: %d destVid: %d\n", srcVid, eSet[tid].dst);
//         printf("vValue: %f weight %f\n", vValues[(srcVid << 1) + 1], eSet[tid].weight);
//         printf("mValueIndex = %d\n", mValueIndex);

        mTransformdMergedMSGValueSet[mValueIndex].destVId = mValueIndex;
        atomicAdd(&mTransformdMergedMSGValueSet[mValueIndex].rank, vValues[(srcVid << 1) + 1] * eSet[tid].weight);
	}
}

hipError_t MSGGenMerge_kernel_exec(PRA_MSG *mTransformdMergedMSGValueSet,
	Vertex *vSet, double *vValues, int numOfEdge, Edge *eSet)
{
	hipError_t err = hipSuccess;

	MSGGenMerge_kernel<<<1, NUMOFGPUCORE>>>(mTransformdMergedMSGValueSet, vSet, vValues, numOfEdge, eSet);
	err = hipGetLastError();

	hipDeviceSynchronize();

	return err;
}